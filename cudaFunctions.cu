#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "definitions.h"
#include <stdio.h>
#include <string.h>

#define NUMOFCONSERVATIVEGROUPS 9
#define NUMOFSEMICONSERVATIVEGROUPS 12
#define NNMOFWIEGHT 4
#define STAR 0
#define COLON 1
#define SIGN 2
#define SPACE 3

__global__ void compareBetweenDNAandRNA(char *DNA, char *RNA, float *result, int size, float *weight);
__device__ void compareTwoChar(float *result, char dnaChar, char rnaChar, int i, float *weight);
__device__ int isBelongToConservativeGroups(char first, char second);
__device__ int isBelongToSemiConservativeGroups(char first, char second);
__device__ int isTheCharsInTheSameGroupAndSameString(const char *str, char first, char second);
__device__ int isTheCharInTheString(const char *str, char ch);

void errorChecking(hipError_t err);
void freeCharacterStringAllocation(hipError_t err, char *characterString);
void freeNumberAllocation(hipError_t err, float *numbers);

/* Compare between the char in DNA to char in RNA */
__global__ void compareBetweenDNAandRNA(char *DNA, char *RNA, float *result, int size, float *weight) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size){
		compareTwoChar(result, DNA[i], RNA[i], i, weight);
	}  
}

/* The function check for each two char what their weight is */
__device__ void compareTwoChar(float *result, char dnaChar, char rnaChar, int i, float *weight){

	result[i] = -(weight[SPACE]);

	if(dnaChar == rnaChar){	
		result[i] = weight[STAR];
	}
	else if (isBelongToConservativeGroups(dnaChar, rnaChar) > 0){
		result[i] = -(weight[COLON]);
	}
	else if (isBelongToSemiConservativeGroups(dnaChar, rnaChar) > 0){
		result[i] = -(weight[SIGN]);
	}
	else{
		result[i] = -(weight[SPACE]);
	}
}

/* The function check if two char belong to conservative groups */
__device__ int isBelongToConservativeGroups(char first, char second){
	
	const char *conservativeGroups[NUMOFCONSERVATIVEGROUPS]= {
		"NDEQ","NEQK","STA","MILV","QHRK","NHQK","FYW","HY","MILF"
	};
	
	// Go over each string in this group and check if two char exist in stirng
	for(int j = 0 ; j < NUMOFCONSERVATIVEGROUPS ; j++){
		if (isTheCharsInTheSameGroupAndSameString(conservativeGroups[j], first, second) != 0) {
				return 1; 
		}	
	}
		
	return 0;
}

/* The function check if two char belong to semi conservative groups */
__device__ int isBelongToSemiConservativeGroups(char first, char second){

	const char *semiConservativeGroups[NUMOFSEMICONSERVATIVEGROUPS]= {
		"SAG","ATV","CSA","SGND","STPA","STNK","NEQHRK","NDEQHK","SNDEQK"," ","HFY","FVLIM"	
	};
	
	// Go over each string in this group and check if two char exist in stirng
	for(int j = 0 ; j < NUMOFSEMICONSERVATIVEGROUPS ; j++){
		if (isTheCharsInTheSameGroupAndSameString(semiConservativeGroups[j], first, second) != 0) {
				return 1; 
		}
	}
	
	return 0;
}

/* The function check if two char exist in the same stirng */
__device__ int isTheCharsInTheSameGroupAndSameString(const char *str, char first, char second){

	int resultFromFirstChar = isTheCharInTheString(str,first);
	int resultFromSecondChar = isTheCharInTheString(str,second);

	if( (resultFromFirstChar != 0) & (resultFromSecondChar != 0)){
		return 1; 
	}

	return 0; 
}

/* The function check if the char exist in string */
__device__ int isTheCharInTheString(const char *str, char ch){

	int i = 0 ;
	
	while(str[i] != '\0'){

		if(str[i] == ch){
			return 1;	
		}
		
		i++;
	}

	return 0;
}

int computeOnGPU(Info info, int offset, float *result){
    
	hipError_t err = hipSuccess;
	
	int sizeOfRna = strlen(info.rna) - 1;
	size_t size = sizeOfRna * sizeof(char);
	size_t sizeForResult = sizeOfRna * sizeof(float);

	float weight[NNMOFWIEGHT] = {info.weightNumber[STAR], info.weightNumber[COLON], info.weightNumber[SIGN],
					info.weightNumber[SPACE]};
  
	// Allocate memory on GPU to copy the data from the host
	char *d_RNA;
	err = hipMalloc((void **)&d_RNA, size);
	errorChecking(err);

	char *d_DNA;
	err = hipMalloc((void **)&d_DNA, size);
	errorChecking(err);

	float *d_Weight;
	err = hipMalloc((void **)&d_Weight, NNMOFWIEGHT*sizeof(float));
	errorChecking(err);

	float *d_C;
	err = hipMalloc((void **)&d_C, sizeForResult);
	errorChecking(err);

	// Copy data from host to the GPU memory
	err = hipMemcpy(d_RNA, info.rna, size, hipMemcpyHostToDevice);
 	errorChecking(err);
	
	err = hipMemcpy(d_DNA, &(info.dna[offset]), size, hipMemcpyHostToDevice);
	errorChecking(err);

	err = hipMemcpy(d_Weight, weight, NNMOFWIEGHT*sizeof(float), hipMemcpyHostToDevice);
	errorChecking(err);
    
	// Launch the Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(sizeOfRna + threadsPerBlock - 1) / threadsPerBlock;
	compareBetweenDNAandRNA<<<blocksPerGrid, threadsPerBlock>>>(d_DNA, d_RNA, d_C, sizeOfRna, d_Weight);
	err = hipGetLastError();
	errorChecking(err);
	
	// Copy the result from GPU to the host memory
	err = hipMemcpy(result, d_C, sizeForResult, hipMemcpyDeviceToHost);
	errorChecking(err);

	// Free allocated memory on GPU
	freeCharacterStringAllocation(err, d_DNA);
	freeCharacterStringAllocation(err, d_RNA);
	freeNumberAllocation(err, d_C);
	freeNumberAllocation(err, d_Weight);
    
  return 0;
}

void errorChecking(hipError_t err){

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void freeCharacterStringAllocation(hipError_t err, char *characterString){

	if (hipFree(characterString) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
      	        exit(EXIT_FAILURE);
	}
}

void freeNumberAllocation(hipError_t err, float *numbers){

	if (hipFree(numbers) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
      	        exit(EXIT_FAILURE);
	}
}

